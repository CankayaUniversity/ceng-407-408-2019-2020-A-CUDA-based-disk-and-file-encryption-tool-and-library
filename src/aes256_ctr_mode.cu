#include "hip/hip_runtime.h"

#include <string.h>
#include "aes.h"
#include <time.h>
#include <stdio.h>
#include <stdint.h>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


__constant__ uint8_t sbox_d[256]= {
		  //0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
		  0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
		  0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
		  0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
		  0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
		  0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
		  0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
		  0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
		  0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
		  0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
		  0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
		  0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
		  0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
		  0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
		  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
		  0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
		  0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };
__constant__ uint8_t rsbox_d[256] = {
		  0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
		  0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
		  0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
		  0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
		  0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
		  0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
		  0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
		  0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
		  0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
		  0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
		  0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
		  0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
		  0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
		  0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
		  0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
		  0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d };
__constant__ uint8_t Rcon_d[11] = {
		  0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };
__constant__ int Nb_d = 4;
__constant__ int Nr_d = 14;
__constant__ int Nk_d = 8;
__constant__ uint32_t ek[60];

// The number of columns comprising a state in AES. This is a constant in AES. Value=4
#define Nb 4

#if defined(AES256) && (AES256 == 1)
    #define Nk 8
    #define Nr 14
#endif





typedef uint8_t state_t[4][4];

static const uint8_t sbox[256] = {
  //0     1    2      3     4    5     6     7      8    9     A      B    C     D     E     F
  0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
  0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
  0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
  0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
  0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
  0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
  0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
  0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
  0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
  0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
  0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
  0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
  0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
  0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
  0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
  0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 };


const uint8_t Rcon[11] = {
  0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

#define getSBoxValue(num) (sbox[(num)]);
#define device_getSBoxValue(num) (sbox_d[(num)]);



inline void cudaDevAssist(hipError_t code, int line, bool abort=true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr,"cudaDevAssistant: %s %d\n", hipGetErrorString(code), line);
		if (abort) exit(code);
	}
}

static void KeyExpansion(uint8_t* RoundKey, const uint8_t* Key)
{
  unsigned i, j, k;
  uint8_t tempa[4]; // Used for the column/row operations

  // The first round key is the key itself.
  for (i = 0; i < Nk; ++i)
  {
    RoundKey[(i * 4) + 0] = Key[(i * 4) + 0];
    RoundKey[(i * 4) + 1] = Key[(i * 4) + 1];
    RoundKey[(i * 4) + 2] = Key[(i * 4) + 2];
    RoundKey[(i * 4) + 3] = Key[(i * 4) + 3];
  }

  // All other round keys are found from the previous round keys.
  for (i = Nk; i < Nb * (Nr + 1); ++i)
  {
    {
      k = (i - 1) * 4;
      tempa[0]=RoundKey[k + 0];
      tempa[1]=RoundKey[k + 1];
      tempa[2]=RoundKey[k + 2];
      tempa[3]=RoundKey[k + 3];

    }

    if (i % Nk == 0)
    {
      // This function shifts the 4 bytes in a word to the left once.
      // [a0,a1,a2,a3] becomes [a1,a2,a3,a0]

      // Function RotWord()
      {
        const uint8_t u8tmp = tempa[0];
        tempa[0] = tempa[1];
        tempa[1] = tempa[2];
        tempa[2] = tempa[3];
        tempa[3] = u8tmp;
      }

      // SubWord() is a function that takes a four-byte input word and
      // applies the S-box to each of the four bytes to produce an output word.

      // Function Subword()
      {
        tempa[0] = getSBoxValue(tempa[0]);
        tempa[1] = getSBoxValue(tempa[1]);
        tempa[2] = getSBoxValue(tempa[2]);
        tempa[3] = getSBoxValue(tempa[3]);
      }

      tempa[0] = tempa[0] ^ Rcon[i/Nk];
    }
#if defined(AES256) && (AES256 == 1)
    if (i % Nk == 4)
    {
      // Function Subword()
      {
        tempa[0] = getSBoxValue(tempa[0]);
        tempa[1] = getSBoxValue(tempa[1]);
        tempa[2] = getSBoxValue(tempa[2]);
        tempa[3] = getSBoxValue(tempa[3]);
      }
    }
#endif
    j = i * 4; k=(i - Nk) * 4;
    RoundKey[j + 0] = RoundKey[k + 0] ^ tempa[0];
    RoundKey[j + 1] = RoundKey[k + 1] ^ tempa[1];
    RoundKey[j + 2] = RoundKey[k + 2] ^ tempa[2];
    RoundKey[j + 3] = RoundKey[k + 3] ^ tempa[3];
  }
}

#if (defined(CTR) && (CTR == 1))
void AES_CTR_iv(struct AES_ctx* ctx, const uint8_t* key, const uint8_t* iv)
{
  KeyExpansion(ctx->RoundKey, key);
  memcpy (ctx->Iv, iv, AES_BLOCKLEN);
}
#endif

// This function adds the round key to state.
// The round key is added to the state by an XOR function.

__device__ void AddRoundKey(uint8_t round, state_t* myState, const uint8_t* RoundKey)
{
  uint8_t i,j;
  //state_t *devState = (state_t*)cipher;

  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      //(cipher)[i*4+j] ^= RoundKey[(round * Nb_d * 4) + (i * Nb_d) + j];
      (*myState)[i][j] ^= RoundKey[(round * Nb_d * 4) + (i * Nb_d) + j];
    	//(cipher)[i*4+j] = 'c';
    }
  }
}


__device__ void SubBytes(state_t* myState)
{
  uint8_t i, j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      //(*devState)[j][i] = getSBoxValue((*devState)[j][i]);
      (*myState)[j][i] = sbox_d[(*myState)[j][i]];
    }
  }
}

// The ShiftRows() function shifts the rows in the state to the left.
// Each row is shifted with different offset.
// Offset = Row number. So the first row is not shifted.

__device__ void ShiftRows(state_t* myState)
{
  uint8_t temp;

  // Rotate first row 1 columns to left
  temp           = (*myState)[0][1];
  (*myState)[0][1] = (*myState)[1][1];
  (*myState)[1][1] = (*myState)[2][1];
  (*myState)[2][1] = (*myState)[3][1];
  (*myState)[3][1] = temp;

  // Rotate second row 2 columns to left
  temp           = (*myState)[0][2];
  (*myState)[0][2] = (*myState)[2][2];
  (*myState)[2][2] = temp;

  temp           = (*myState)[1][2];
  (*myState)[1][2] = (*myState)[3][2];
  (*myState)[3][2] = temp;

  // Rotate third row 3 columns to left
  temp           = (*myState)[0][3];
  (*myState)[0][3] = (*myState)[3][3];
  (*myState)[3][3] = (*myState)[2][3];
  (*myState)[2][3] = (*myState)[1][3];
  (*myState)[1][3] = temp;
}


__device__ uint8_t xtime(uint8_t x)
{
  return ((x<<1) ^ (((x>>7) & 1) * 0x1b));
}

// MixColumns function mixes the columns of the state matrix

__device__ void MixColumns(state_t* myState)
{
	uint8_t i;
	uint8_t Tmp, Tm, t;
	for (i = 0; i < 4; ++i)
	{
		t   = (*myState)[i][0];
		Tmp = (*myState)[i][0] ^ (*myState)[i][1] ^ (*myState)[i][2] ^ (*myState)[i][3] ;
		Tm  = (*myState)[i][0] ^ (*myState)[i][1] ;
		Tm = xtime(Tm); (*myState)[i][0] ^= Tm ^ Tmp ;
		Tm  = (*myState)[i][1] ^ (*myState)[i][2] ;
		Tm = xtime(Tm); (*myState)[i][1] ^= Tm ^ Tmp ;
		Tm  = (*myState)[i][2] ^ (*myState)[i][3] ;
		Tm = xtime(Tm); (*myState)[i][2] ^= Tm ^ Tmp ;
		Tm  = (*myState)[i][3] ^ t ;
		Tm = xtime(Tm); (*myState)[i][3] ^= Tm ^ Tmp ;
	}
}

// GPUCipher is the main function that encrypts the PlainText.
__global__ void GPUCipher(state_t* devState, const uint8_t* RoundKey, uint8_t* plain_text_d, state_t* myIv, int count) // HT
{
	int id = threadIdx.x;
	uint8_t round = 0;
	unsigned i;

	//uint8_t *myIv= (uint8_t *) (devState); // HT
	//state_t *myState = (state_t *) (myIv); // HT

//	for(int x = 0; x < 4; x++){
//		for(int y = 0; y < 4; y++){
//			(*myState)[x][y] = (*devState)[x][y]; // HT
//			//printf("My state: %d " ,myState[x][y]);
//		}
//	}
	AddRoundKey(0, devState, RoundKey); //devState -> myState
	for (round = 1; ; ++round)
	{
		SubBytes(devState); // getSBoxValue !!! // HT
		ShiftRows(devState); // HT
		if (round == Nr_d) {
		  break;
		}
		MixColumns(devState); // HT
		AddRoundKey(round, devState, RoundKey); // HT
	}

	 //Add round key to last round
	AddRoundKey(Nr_d, devState, RoundKey); // HT

	for(int a=0; a < id; ++a){
		for ( i = 0 ; i < 16; ++i)
		{
			/* inc will overflow */
			if (((uint8_t *)myIv)[i] == 255)
			{
				((uint8_t *)myIv)[i] = 0;
				continue;
			}
			//printf("%d", (int*)myIv[i]);
			((uint8_t *)myIv)[i] += 1;
			break;
		}
		plain_text_d[(id * 16) + i] = plain_text_d[(id * 16) + i] ^ ((uint8_t *)devState)[i];
//		//printf("Plaint text: %s ",(char*)plain_text_d);
	}
}
static void AddRoundKeyCPU(uint8_t round, state_t* state, const uint8_t* RoundKey)
{
  uint8_t i,j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
      (*state)[i][j] ^= RoundKey[(round * Nb * 4) + (i * Nb) + j];
    }
  }
}
static void SubBytesCPU(state_t* state)
{
  uint8_t i, j;
  for (i = 0; i < 4; ++i)
  {
    for (j = 0; j < 4; ++j)
    {
//      (*state)[j][i] = getSBoxValue((*state)[j][i]);
    	(*state)[j][i] = sbox[((*state)[j][i])];
    }
  }
}
static void ShiftRowsCPU(state_t* state)
{
  uint8_t temp;

  // Rotate first row 1 columns to left
  temp           = (*state)[0][1];
  (*state)[0][1] = (*state)[1][1];
  (*state)[1][1] = (*state)[2][1];
  (*state)[2][1] = (*state)[3][1];
  (*state)[3][1] = temp;

  // Rotate second row 2 columns to left
  temp           = (*state)[0][2];
  (*state)[0][2] = (*state)[2][2];
  (*state)[2][2] = temp;

  temp           = (*state)[1][2];
  (*state)[1][2] = (*state)[3][2];
  (*state)[3][2] = temp;

  // Rotate third row 3 columns to left
  temp           = (*state)[0][3];
  (*state)[0][3] = (*state)[3][3];
  (*state)[3][3] = (*state)[2][3];
  (*state)[2][3] = (*state)[1][3];
  (*state)[1][3] = temp;
}
static uint8_t xtimeCPU(uint8_t x)
{
  return ((x<<1) ^ (((x>>7) & 1) * 0x1b));
}
static void MixColumnsCPU(state_t* state)
{
  uint8_t i;
  uint8_t Tmp, Tm, t;
  for (i = 0; i < 4; ++i)
  {
    t   = (*state)[i][0];
    Tmp = (*state)[i][0] ^ (*state)[i][1] ^ (*state)[i][2] ^ (*state)[i][3] ;
    Tm  = (*state)[i][0] ^ (*state)[i][1] ; Tm = xtimeCPU(Tm);  (*state)[i][0] ^= Tm ^ Tmp ;
    Tm  = (*state)[i][1] ^ (*state)[i][2] ; Tm = xtimeCPU(Tm);  (*state)[i][1] ^= Tm ^ Tmp ;
    Tm  = (*state)[i][2] ^ (*state)[i][3] ; Tm = xtimeCPU(Tm);  (*state)[i][2] ^= Tm ^ Tmp ;
    Tm  = (*state)[i][3] ^ t ;              Tm = xtimeCPU(Tm);  (*state)[i][3] ^= Tm ^ Tmp ;
  }
}

static void CPUCipher(state_t* state, const uint8_t* RoundKey)
{
  uint8_t round = 0;

  // Add the First round key to the state before starting the rounds.
  AddRoundKeyCPU(0, state, RoundKey);

  // There will be Nr rounds.
  // The first Nr-1 rounds are identical.
  // These Nr rounds are executed in the loop below.
  // Last one without MixColumns()
  for (round = 1; ; ++round)
  {
    SubBytesCPU(state);
    ShiftRowsCPU(state);
    if (round == Nr) {
      break;
    }
    MixColumnsCPU(state);
    AddRoundKeyCPU(round, state, RoundKey);
  }
  // Add round key to last round
  AddRoundKeyCPU(Nr, state, RoundKey);
}



#if defined(CTR) && (CTR == 1)

/* Symmetrical operation: same function for encrypting as for decrypting. */
void AES_CTR_encryption(struct AES_ctx* ctx, uint8_t* buf, uint32_t length, int block_count , int count)
{

	uint8_t buffer[AES_BLOCKLEN];
	state_t *devState = NULL;
	uint8_t *roundKey_d = NULL;
	uint8_t *plain_text_d = NULL;
	state_t *myIv = NULL; // HT
	uint8_t *plain_text_h = NULL;
	uint8_t *buffer2;
	//uint8_t *buffer_d = NULL;
	unsigned i;
	int bi;
	if(count > 524288){ //GPU Encryption
		for (i = 0, bi = AES_BLOCKLEN; i < length; ++i, ++bi)
		{
			if (bi == AES_BLOCKLEN) /* we need to regen xor compliment in buffer */
			{

				//printf("\nENC: %s",(char*) buf);

				memcpy(buffer, ctx->Iv, AES_BLOCKLEN);
				//printf("\nbuffer: %s",(char*) buffer);
				hipSetDevice(1);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nk_d), &Nk, sizeof(int), 0, hipMemcpyHostToDevice), 535, true);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nr_d), &Nr, sizeof(int), 0, hipMemcpyHostToDevice), 543, true);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nb_d), &Nb, sizeof(int), 0, hipMemcpyHostToDevice), 903, true);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(sbox_d), &sbox, 256*sizeof(uint8_t), 0, hipMemcpyHostToDevice), 920, true);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(rsbox_d), &rsbox, 256*sizeof(uint8_t), 0, hipMemcpyHostToDevice), 921, true);
	//			cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Rcon_d), &Rcon, 11*sizeof(uint8_t), 0, hipMemcpyHostToDevice), 922, true);
				//hipDeviceSynchronize();
				cudaDevAssist(hipMalloc((void**)&devState, 16*sizeof(uint8_t)), 452, true);
				cudaDevAssist(hipMalloc((void**)&roundKey_d, 240*sizeof(uint8_t)), 453, true);
				cudaDevAssist(hipMalloc((void**)&plain_text_d, count*sizeof(uint8_t)),446, true);
				cudaDevAssist(hipMalloc((void**)&myIv, 16*sizeof(uint8_t)),447, true);

				// Time starting
				cudaDevAssist(hipMemcpy(devState, &buffer, 16*sizeof(uint8_t), hipMemcpyHostToDevice), 455, true);
				cudaDevAssist(hipMemcpy(myIv, &buffer, 16*sizeof(uint8_t), hipMemcpyHostToDevice), 455, true);
				cudaDevAssist(hipMemcpy(roundKey_d, ctx->RoundKey, 240*sizeof(uint8_t), hipMemcpyHostToDevice), 456, true);
				cudaDevAssist(hipMemcpy(plain_text_d, buf, count*sizeof(uint8_t), hipMemcpyHostToDevice), 457, true);
				//printf("\nENC2: %s",(char*)roundKey_d);
				//cudaDevAssist(hipMemcpy(buffer_d, buffer, textLength*sizeof(uint8_t), hipMemcpyHostToDevice), 457, true);
				cudaDevAssist(hipDeviceSynchronize(), 268, true);
				//cudaCipher<<<1,1>>>(devState,roundKey_d,buffer_d);
				GPUCipher<<<1,block_count>>>(devState,roundKey_d, plain_text_d,myIv,count);

				buffer2 = (uint8_t*)malloc(16*sizeof(uint8_t));
				plain_text_h = (uint8_t*)malloc(count*sizeof(uint8_t));
				cudaDevAssist(hipMemcpy(buffer2, devState, 16*sizeof(uint8_t), hipMemcpyDeviceToHost), 462, true);
				//cudaDevAssist(hipMemcpy(plain_text_h, plain_text_d, count*sizeof(uint8_t), hipMemcpyDeviceToHost), 463, true);
				//Time ending

				  /* Increment Iv and handle overflow */
//				for (bi = (AES_BLOCKLEN - 1); bi >= 0; --bi) //Önce hesapla sonra gönder.
//				{
//					/* inc will overflow */
//					if (ctx->Iv[bi] == 255)
//					{
//						ctx->Iv[bi] = 0;
//						continue;
//					}
//					ctx->Iv[bi] += 1;
//					break;
//				}
//				bi = 0;
			}

			//buf[i] = (buf[i] ^ buffer2[bi]);


		}
		hipFree(devState);
		hipFree(roundKey_d);
		hipFree(plain_text_d);
		hipFree(myIv);
		memcpy(buf, plain_text_h,count);
	}
	else{ //CPU Encryption
		for (i = 0, bi = AES_BLOCKLEN; i < length; ++i, ++bi)
		  {
		    if (bi == AES_BLOCKLEN) /* we need to regen xor compliment in buffer */
		    {

				  memcpy(buffer, ctx->Iv, AES_BLOCKLEN);
				  CPUCipher((state_t*)buffer,ctx->RoundKey);

				  /* Increment Iv and handle overflow */
				  for (bi = (AES_BLOCKLEN - 1); bi >= 0; --bi)
				  {
					/* inc will overflow */
						if (ctx->Iv[bi] == 255)
						{
						  ctx->Iv[bi] = 0;
						  continue;
						}
						ctx->Iv[bi] += 1; // 12A3  + 1 12A4 + 1
						break;
				  }
				  bi = 0;
		    }

		    buf[i] = (buf[i] ^ buffer[bi]); // buf = plain text, buffer = Iv
		  }
	}
}

#endif // #if defined(CTR) && (CTR == 1)
int main(int argc, const char * argv[])
{
	struct AES_ctx ctx;
	clock_t c_start, c_stop;

    uint8_t iv[16] = { 0xf0, 0xf1, 0xf2, 0xf3, 0xf4, 0xf5, 0xf6, 0xf7, 0xf8, 0xf9, 0xfa, 0xfb, 0xfc, 0xfd, 0xfe, 0xff };
    FILE *infile, *outfile, *keyfile;


	printf("Enter the path of input file");
	scanf("%s", argv[0]);
	printf("Enter the path of 32-bit key file");
	scanf("%s", argv[2]);






//    if(size % 16 != 0){
//    	int addition_part = 16 - (size % 16);
//		in = (uint8_t*)realloc(in,5*sizeof(int));
//		for (int i = size; i <= size + addition_part; ++i)
//			in[i+1] = 0;
//    }


    infile = fopen(argv[0], "r");
    fseek(infile, 0, SEEK_END);
	int size = ftell(infile);
	fseek(infile, 0, SEEK_SET);
	uint8_t in[size];
    int count = 0;
    for(int i = 0; i<size; ++i){
    	fread(&in, sizeof(char), size, infile);
    	count++;
    }
	fclose(infile);
	printf("\nData read from file: %s\n", in);

    keyfile = fopen(argv[2], "r");
	uint8_t key[32];
	fread(&key, sizeof(char), 32, keyfile);
	fclose(keyfile);
	printf("\nKeyfile: %s \n", key);


	int block_count = (count / 16) + (count % 16 == 0 ? 0 : 1); // 16 - count%16 kadar 0 eklenicek.


	printf("File size: %d bytes\n", size);

	outfile = fopen("/home/emre/cuda-workspace/AES256_CTR/src/output.txt", "w");
	// /home/emre/Desktop/Test_files/1kb.txt   /home/emre/cuda-workspace/AES256_CTR/src/key.txt
	int breaking_point = 524288; //This is the breaking point of our project. If file size is less than 512 kb, it runs on CPU, if it is larger than 512 kb, it runs on GPU.
	if (count >= breaking_point){

		printf("GPU initiliaze\n");
		printf("Data read from file: %s\n", in);
		AES_CTR_iv(&ctx, key, iv);
		c_start = clock();
		printf("Elements read: %d", count);
		printf("\nBlock Count: %d", block_count);
		AES_CTR_encryption(&ctx, in, strlen((char*)in),block_count,count);
		printf("\nENC: %s",(char*) in);
		fwrite(&in, sizeof(char), count, outfile);
		c_stop = clock();
		float diff = (((float)c_stop - (float)c_start) / CLOCKS_PER_SEC ) * 1000;
		printf("\nDone - Time taken on GPU: %f ms\n", diff);

	}
	else{
		printf("CPU initiliaze\n");


		//printf("keyfile: %s\n", key);
		printf("Elements read: %d\n", count);

		AES_CTR_iv(&ctx, key, iv);
		uint8_t Input[256];
		//uint8_t in2[size];
		printf("\nEnc:");
		c_start = clock();
		for(int i = 256, a=0; i<256+size;i+=256,a+=256){

			memcpy(Input, in+a,256);

			AES_CTR_encryption(&ctx, Input, strlen((char*)Input),block_count,count);
			printf("%s\n",(char*) Input); // don't use this string as an input
			fwrite(&Input, sizeof(char), 256, outfile);
		}
		c_stop = clock();
		float diff = (((float)c_stop - (float)c_start) / CLOCKS_PER_SEC ) * 1000;
		printf("\nDone - Time taken on CPU: %f ms\n", diff);
	}
	fclose(outfile);


    return 0;
}
